#include "hip/hip_runtime.h"
#include <GL/glew.h> 
#include <GL/gl.h> 
#include <cuda_gl_interop.h> 
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

extern "C" void sort_pixels(size_t num_pixels);
extern "C" void register_buffer(GLuint buffer);

static GLuint  bufferObj;
static hipGraphicsResource *resource;

struct sort_functor
{
	__host__ __device__
	bool operator()(float4 left, float4 right) const
	{
		return (left.z < right.z);
	}
};

extern "C"
void sort_pixels(size_t num_pixels) {
	hipGraphicsMapResources(1, &resource, NULL);
	float4* devPtr;
	size_t  size;

	hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, resource);
	thrust::device_ptr<float4> tptr = thrust::device_pointer_cast(devPtr);
	thrust::sort(tptr, tptr + (num_pixels), sort_functor());
	hipGraphicsUnmapResources(1, &resource, NULL);
}

extern "C"
void register_buffer(GLuint buffer)
{
	bufferObj = buffer;
	hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone);
}
